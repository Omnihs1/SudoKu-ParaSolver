#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>
#include "Board.hpp"
#include "parallelsudoku.cuh"
#include "CycleTimer.h"
#define UPDIV(n, d) (((n)+(d)-1)/(d))
#define NDEBUG 1
const int threadsPerBlock = 512;

// function to examine if there are conflicts or not if cell [row][col] is num
__device__
bool noConflicts(int matrix[boardSize * boardSize], int row, int col, int num) {
    if (num <= 0 || num > boardSize) return false;
    for (int i = 0; i < boardSize; i++) {
        if (i == row)   continue;
        if (matrix[i * boardSize + col] == num) {
            return false;
        }
    }
    
    for (int j = 0; j < boardSize; j++) {
        if (j == col)   continue;
        if (matrix[row * boardSize + j] == num) {
            return false;
        }
    }
    
    for (int i = 0; i < box_size; i++) {
        for (int j = 0; j < box_size; j++) {
            int mat_row = (row/box_size)*box_size + i;
            int mat_col = (col/box_size)*box_size + j;
            if (mat_row == row && mat_col == col)   continue;
            if (matrix[mat_row * boardSize + mat_col] == num) {
                return false;
            }
        }
    }
    return true;
}

// find the next empty cell index 
__device__
int findNextEmptyCellIndex(int matrix[boardSize*boardSize], int start) {
    int i;
    for (i = start; i < boardSize*boardSize; i++) {
        if (matrix[i] == 0) {
            return i;
        }
    }
    return i;
}

// the kernel that solves sudoku problem on each board
// each thread works on a board in the boards array
__global__
void SolvingKernel(int* boards, int boardCnt, int* solution, int numThreads, int *finished) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("\n");
    int localBoard[boardSize*boardSize];
    for (int idx = tidx; (idx < boardCnt) && (*finished == 0); idx += numThreads) {
        int emptyCnt = 0;
        int emptyIndex[boardSize*boardSize];
        int start = idx * boardSize * boardSize;
        for (int i = start; i < (idx+1) * boardSize * boardSize; i++) {
            localBoard[i-start] = boards[i];
            if (!localBoard[i-start]) {
                emptyIndex[emptyCnt] = i-start;
                emptyCnt++;
            }
        }
#ifndef NDEBUG
        printf("\n%d, %d\n", idx, emptyCnt);
#endif
        // backtracking algorithm
        int depth = 0;
        while (depth >= 0 && depth < emptyCnt) {
            int next = emptyIndex[depth];
            int row = next / boardSize;
            int col = next % boardSize;
            localBoard[next]++;
            if (noConflicts(localBoard, row, col, localBoard[next])) depth++;
            else if (localBoard[next] >= boardSize) {
                localBoard[next] = 0;
                depth--;
            }
        }
        if (depth == emptyCnt) {
            // solved board found 
            *finished = 1;
            // copy board to output
            // printf("\n%d\n", idx);
            memcpy(solution, localBoard, boardSize*boardSize*sizeof(int));
            break;
        }
    }
}



// kernel that generates new boards from previous ones
// call this kernel multiple times to have 
__global__
void BoardGenerationKernel(int* prev_boards, int* board_num, int prev_board_num, int* new_boards, int numThreads) {
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int* localBoard = (int*) malloc(sizeof(int)*boardSize*boardSize);

    if (!prev_board_num) prev_board_num++;
    for (int idx = tidx; idx < prev_board_num; idx+=numThreads) {
        int start = idx * boardSize * boardSize;
        for (int i = start; i < (idx+1) * boardSize * boardSize; i++) {
            __syncthreads();
            localBoard[i-start] = prev_boards[i];
        }
        int emptyIdx = findNextEmptyCellIndex(localBoard, 0);
        if (emptyIdx == boardSize*boardSize)  return;
        for (int k = 1; k <= boardSize; k++) {
            localBoard[emptyIdx] = k;
            if (noConflicts(localBoard, emptyIdx/boardSize, emptyIdx%boardSize, k)) {
                int offset = atomicAdd(board_num, 1);
                for (int ii = 0; ii < boardSize*boardSize; ii++) {
                    __syncthreads();
                    new_boards[boardSize*boardSize*offset+ii] = localBoard[ii];
                    __syncthreads();
                }
            }
        }
    }
}

void 
BoardGenerator(int* prev_boards, int* prev_board_num, int* new_boards, int DEPTH) {
    int i;
    int num = 1;
    for (i = 0; i < DEPTH; i++) {
        int block = UPDIV(num, threadsPerBlock);
        hipMemset(prev_board_num, 0, sizeof(int));
        if(i%2 ==  0){
          BoardGenerationKernel<<<block, threadsPerBlock>>>(prev_boards, prev_board_num, num, new_boards, block*threadsPerBlock);
        }
        else{
          int *temp = new_boards;
          prev_boards = temp;
          BoardGenerationKernel<<<block, threadsPerBlock>>>(prev_boards, prev_board_num, num, new_boards, block*threadsPerBlock);
        }
        hipMemcpy(&num, prev_board_num, sizeof(int), hipMemcpyDeviceToHost);
        printf("total boards after an iteration %d: %d \n", i + 1, num);
    }
#ifndef NDEBUG
    int memSize = pow(2, 26);
    int *board = (int*)malloc(memSize * sizeof(int));
    hipMemcpy(board, new_boards, memSize * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < num; i++){
      for(int j = 0; j < 81; j++){
        printf("%5d", board[i*81+j]);
      }
      printf("\n");
    }
#endif
}

void 
cudaSudokuSolver(int* boards, int board_num, int* solution) {
    int block = UPDIV(board_num, threadsPerBlock);
    int *finished;
    hipMalloc(&finished, sizeof(int));
    hipMemset(finished, 0, sizeof(int));
    cout << "Block: " << block << ", " << "threadsPerBlock: " << threadsPerBlock << endl;
    SolvingKernel<<<block, threadsPerBlock>>>(boards, board_num, solution, block*threadsPerBlock, finished);
    hipDeviceSynchronize();
}